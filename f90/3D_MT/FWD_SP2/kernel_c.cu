#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <complex.h>

// simple kernel function that converts double vectors to single
__global__ void real64to32(const double *in, float *out, const int N)
{
    // a position every 64 bits
    // int pos = blockDim.x * blockIdx.x + threadIdx.x;
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	float sing = __double2float_rn(in[pos]);
	out[pos] = sing;
    }
}

// simple kernel function that converts single vectors to double
__global__ void real32to64(const uint32_t *in, uint64_t *out, const int N)
{
    // a position every 32 bits
    // int pos = blockDim.x * blockIdx.x + threadIdx.x;
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	// last sign bit (1)
	uint64_t s = in[pos] & 0x80000000;
	// exponent bits (8)
	uint64_t e = ((in[pos] & 0x7f800000) >> 23);
	e = e + 896;
	// mantissa bits (23)
	uint64_t m = in[pos] & 0x7fffff;
	// double through bitwise or
	uint64_t doub = (s<<32) | (e<<52) | (m<<29);
        // a new position every 64 bits
	out[pos] = doub;
    }
 }

__global__ void hada_real(const double *ina, const double *inb, double *out, const int N)
{
    //hadamard multiplication, as real 
    // c = a * b
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	out[pos] = ina[pos]*inb[pos];
    }
 }

__global__ void hada_cmplx(const double *ina, const double *inb, double *out, const int N)
{
    //hadamard multiplication, as complex
    //(a + bi) * (c + di) = (ac -bd) + (ad + bc)i
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
        if ( pos%2 == 0) // 0 2 4 
        {
	        out[pos] = ina[pos]*inb[pos]-ina[pos+1]*inb[pos+1];
        }
        else // 1 3 5 
        {
	        out[pos] = ina[pos-1]*inb[pos]+ina[pos]*inb[pos-1];
        }
    }
 }

__global__ void xpby_real(const double *x, const double b, double *y, const int N)
{
    // CUDA kernel implementing xpby:
    // y = x + b*y
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	        y[pos] = x[pos]+b*y[pos];
    }
}

__global__ void xpby_cmplx(const double *x, double br, double bi, double *y, const int N)
{
    // CUDA kernel implementing xpby:
    // y = x + b*y
    // complex makes this a little tricky - need a temp variable
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // lower 64bit --> real
	    double lower = x[pos] + br*y[pos] - bi*y[pos+1];
        // higher 64bit --> imag
	    y[pos+1] = x[pos+1] + br*y[pos+1] + bi*y[pos];
        // and copy them back
        y[pos] = lower;
    }
}

__global__ void p_update_real(const double *v, const double *r, const double beta, const double omega, double *p, const int N)
{
    // CUDA kernel implementing p update:
    // p = r + beta * (p - omega*v)
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	    p[pos] = r[pos] + beta*(p[pos] - omega*v[pos]);
    }
}

__global__ void p_update_cmplx(const double *r, const double *v, const double br, const double bi, const double wr, const double wi, double *p, const int N)
{
    // CUDA kernel implementing p update:
    // p = r + beta * (p - omega*v)
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // lower 64bit --> real p - omega*v
	    double lower = p[pos] - wr*v[pos] + wi*v[pos+1];
        // higher 64bit --> imag p - omega*v
	    double higher = p[pos+1] - wr*v[pos+1] - wi*v[pos];
        // lower 64bit --> real r + beta * (p - omega*v)
	    p[pos] = r[pos] + br*lower - bi*higher;
        // higher 64bit --> imag r + beta * (p - omega*v)
	    p[pos+1] = r[pos+1] + br*higher + bi*lower;
    }
}

__global__ void x_update_cmplx(const double *p, const double *s, const double ar, const double ai, const double wr, const double wi, double *x, const int N)
{
    // CUDA kernel implementing p update:
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // x = x + alpha * ph + omega * sh
        // lower 64bit --> real x + alpha*ph
	    x[pos] = x[pos] + ar*p[pos] - ai*p[pos+1] + wr*s[pos] - wi*s[pos];
        // higher 64bit --> imag x + alpha*ph
	    x[pos+1]= x[pos+1] + ar*p[pos+1] + ai*p[pos] + wr*s[pos+1] + wi*s[pos];
    }
}

__global__ void reduce_real(double *a, double s, const int N)
{
    // CUDA kernel implementing reduce s = sum(a)
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    int number_of_threads = (blockDim.x*blockDim.y);
    int step = 1; //initial step
    while (number_of_threads > 0)
    {
        if (pos < number_of_threads) 
        {
            const int left = pos * step*2; 
            const int right = left + step;
            a[left] += a[right];
        }
        step <<= 1;
        number_of_threads >>= 1;
        __syncthreads();
    }
}

// function to wait for sometime
void sleep(int seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000 * seconds;
    // Storing start time
    clock_t start_time = clock();
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

// function called from main fortran program
// single to double conversion
extern "C" void kernelc_s2d(const uint32_t *a_d, uint64_t *b_d, int Np)
{
    //uint32_t  *a_d;  // declare GPU vector double (but stored as uint)
    //uint64_t  *b_d;  // declare GPU vector float
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    real32to64<<< grids, blocks >>>( a_d, b_d, N);

    return;
}

// function called from main fortran program
// double to single conversion
extern "C" void kernelc_d2s(const double *a_d, float *b_d, int Np)
{
    //double  *a_d;  // declare GPU vector double 
    //float  *b_d;  // declare GPU vector float
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    real64to32<<< grids, blocks >>>( a_d, b_d, N);

    return;
}

// function called from main fortran program
// hadamard multiply real version
extern "C" void kernelc_hadar(const double *a_d, const double *b_d, double *c_d, int Np)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
    //double  *c_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    hada_real<<< grids, blocks >>>( a_d, b_d, c_d, N);

    return;
}

// function called from main fortran program
// hadamard multiply complex version
extern "C" void kernelc_hadac(const double *a_d, const double *b_d, double *c_d, int Np)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
    //double  *c_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    hada_cmplx<<< grids, blocks >>>( a_d, b_d, c_d, N);

    return;
}

// function called from main fortran program
// xpby complex version
extern "C" void kernelc_xpbyc(const double *x_d, double _Complex b, double *y_d, int Np)
{
    //double  *x_d;  // declare GPU vector double 
    //double  *y_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double br = creal(b);
    // get lower 64 bit
    double bi = cimag(b);
    // call function on GPU
    xpby_cmplx<<< grids, blocks >>>( x_d, br, bi, y_d, N);

    return;
}

// function called from main fortran program
// update p complex version
extern "C" void kernelc_update_pc(const double *r_d, const double *v_d, double _Complex beta, double _Complex omega, double *p_d, int Np)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double br = creal(beta);
    // get lower 64 bit
    double bi = cimag(beta);
    // get upper 64 bit
    double wr = creal(omega);
    // get lower 64 bit
    double wi = cimag(omega);
    // call function on GPU
    p_update_cmplx<<< grids, blocks >>>( r_d, v_d, br, bi, wr, wi, p_d, N);

    return;
}

// function called from main fortran program
// update x complex version
extern "C" void kernelc_update_xc(const double *ph_d, const double *sh_d, double _Complex alpha, double _Complex omega, double *x_d, int Np)
{
    //double  *ph_d;  // declare GPU vector double 
    //double  *sh_d;  // declare GPU vector double
    //double  *x_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double ar = creal(alpha);
    // get lower 64 bit
    double ai = cimag(alpha);
    // get upper 64 bit
    double wr = creal(omega);
    // get lower 64 bit
    double wi = cimag(omega);
    // call function on GPU
    x_update_cmplx<<< grids, blocks >>>( ph_d, sh_d, ar, ai, wr, wi, x_d, N);

    return;
}

// function called from main fortran program 
extern "C" int kernelc_hookCtx(int dev_idx)
{
    nvmlReturn_t result;
    hipError_t err;
    hipCtx_t thisCtx;
    nvmlDevice_t device;
    nvmlPciInfo_t pci_bus;
    nvmlUtilization_t usage;
    unsigned int Ndevice;
    // unsigned int i;
    // unsigned int nProc = 32;
    // nvmlProcessInfo_t pInfo[nProc];
    char device_name[NVML_DEVICE_NAME_BUFFER_SIZE];

    // First initialize NVML library
    result = nvmlInit();
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        return 1;
    }
    result = nvmlDeviceGetCount(&Ndevice);
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        return 1;
    }
    if (dev_idx + 1 > Ndevice) 
    { 
        printf("Error: not enough devices detected \n");
        printf("required device idx = %i, Ndevice = %u \n", dev_idx, Ndevice);
        goto Error;
    }
    err = hipSetDevice(dev_idx);
	if( err != hipSuccess )
    {
        printf("Failed to set device %i: %u \n", dev_idx, err);
        goto Error;
    }
    while(true)
    {
        // Query for device handle to perform operations on a device
        // You can also query device handle by other features like:
        // nvmlDeviceGetHandleBySerial
        // nvmlDeviceGetHandleByPciBusId
        result = nvmlDeviceGetHandleByIndex(dev_idx, &device);
        if (NVML_SUCCESS != result)
        { 
            printf("Failed to get handle for device %i: %s\n", dev_idx, nvmlErrorString(result));
            goto Error;
        }
        // now get the current context (if any)
        hipCtxGetCurrent(&thisCtx);
        if(thisCtx == NULL) // first call to this device
        {
            hipCtxCreate(&thisCtx, 0, dev_idx);
            result = nvmlDeviceGetName(device, device_name, NVML_DEVICE_NAME_BUFFER_SIZE);
            if (NVML_SUCCESS != result)
            { 
                printf("Failed to get name of device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            // pci.busId is very useful to know which device physically 
            // you're talking to
            // Using PCI identifier you can also match nvmlDevice 
            // handle to CUDA device.
            result = nvmlDeviceGetPciInfo(device, &pci_bus);
            if (NVML_SUCCESS != result)
            { 
                printf("Failed to get pci info for device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            break;// just go ahead to initialize the case
        }
        else// see if this device is available
        {
            hipCtxSetCurrent(thisCtx);
            result = nvmlDeviceGetUtilizationRates( device, &usage );
            if (NVML_SUCCESS != result)
            {
                printf("Failed to get usage for device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            if (usage.memory+usage.gpu < 15)
            {
                printf(" # Dev Status  : GPU-util=%u GPU-mem=%u \n", usage.gpu, usage.memory );
                result = nvmlDeviceGetName(device, device_name, NVML_DEVICE_NAME_BUFFER_SIZE);
                if (NVML_SUCCESS != result)
                { 
                    printf("Failed to get name of device %i: %s\n", dev_idx, nvmlErrorString(result));
                    goto Error;
                }
                // pci.busId is very useful to know which device physically 
                // you're talking to
                // Using PCI identifier you can also match nvmlDevice 
                // handle to CUDA device.
                result = nvmlDeviceGetPciInfo(device, &pci_bus);
                if (NVML_SUCCESS != result)
                { 
                    printf("Failed to get pci info for device %i: %s\n", dev_idx, nvmlErrorString(result));
                    goto Error;
                }
                break;
            }
        }
        sleep(0.05);
    }
    printf(" # Dev Selected:  %i. %s [%s]\n", dev_idx, device_name, pci_bus.busId);
    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    return 0;
Error:
    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    return 1;
}

// externel function to tell the number of GPU devices
extern "C" int kernelc_getDevNum()
{
    nvmlReturn_t result;
    unsigned int nGPU;
    int          Ndevice;

    // First initialize NVML library
    result = nvmlInit();
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        return -1;
    }

    // number of devices
    result = nvmlDeviceGetCount(&nGPU);
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to query device count: %s\n", nvmlErrorString(result));
        goto Error;
    }
    // go silent
    // printf("Found %u device%s\n", Ndevice, Ndevice != 1 ? "s" : " ");

    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    Ndevice = nGPU;
    return Ndevice;
Error:
    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    Ndevice = 0;
    return -1;
}
